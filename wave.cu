#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/*
    This example demonstrates how to use the Cuda OpenGL bindings to
    dynamically modify a vertex buffer using a Cuda kernel.

    The steps are:
    1. Create an empty vertex buffer object (VBO)
    2. Register the VBO with Cuda
    3. Map the VBO for writing from Cuda
    4. Run Cuda kernel to modify the vertex positions
    5. Unmap the VBO
    6. Render the results using OpenGL

    Host code
*/

#define GLEW_STATIC
#define FREEGLUT_STATIC

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <cstdlib>
#include <ctime>
#include <random>
#include <chrono>
using namespace std::chrono;

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include <helper_gl.h>
#if defined (__APPLE__) || defined(MACOSX)
  #pragma clang diagnostic ignored "-Wdeprecated-declarations"
  #include <GLUT/glut.h>
  #ifndef glutCloseFunc
  #define glutCloseFunc glutWMCloseFunc
  #endif
#else
#include <GL/freeglut.h>
#endif

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

#include <hip/hip_vector_types.h>

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     10 //ms

////////////////////////////////////////////////////////////////////////////////
// constants
const unsigned int window_width  = 512;
const unsigned int window_height = 512;

const unsigned int mesh_width    = 256;
const unsigned int mesh_height   = 256;

// vbo variables
GLuint vbo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;
/*
//20/10/19
GLuint vbo2;
struct hipGraphicsResource *cuda_vbo_resource_2;
void *d_vbo_buffer_2 = NULL;
*/

//22/10/19 test
float4 *h_offsets;
float4 *d_offsets;
//void *offsetsAutoTestH = NULL;
void *offsetsAutoTestD = NULL;

float g_fAnim = 0.0;
float UnitOfChangeOnY = 0.0f;// 19/10/19 test -UI
float UnitOfChangeOnX = 0.0f;

//int jitterAmmountInt = 0;//20/10/19
float jitterAmmountFloat1 = 0.0f;//20/10/19
float jitterAmmountFloat2 = 0.0f;
float jitterAmmountFloat3 = 0.0f;
bool jitter = false;

bool origionalJitter = false;
float jitterAmmountFloatOrigional = 0.0f;
//bool exitTest = false;

//21/10/19 game
//float fallingDistence = 0.0f;
bool falling = false;
float horizontalChange = 0.0f;
float randomHeightTop = 0.0f;
float randomHeightBottom = 0.0f;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

StopWatchInterface *timer = NULL;

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

int *pArgc = NULL;
char **pArgv = NULL;

#define MAX(a,b) ((a > b) ? a : b)

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
bool runTest(int argc, char **argv, char *ref_file);
void cleanup();

// GL functionality
bool initGL(int *argc, char **argv);
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags);

//20/10/19
void createVBO2(GLuint *vbo, struct hipGraphicsResource **vbo_res,
	unsigned int vbo_res_flags);

void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res);

// rendering callbacks
void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);

// Cuda functionality
void runCuda(struct hipGraphicsResource **vbo_resource, float4 **Hoffsets, float4 **Doffsets);//, float4 * h_offsets, float4 * d_offsets);//, struct hipGraphicsResource **vbo_resource_2);//20/10/19 test
void runAutoTest(int devID, char **argv, char *ref_file);
void checkResultCuda(int argc, char **argv, const GLuint &vbo);

const char *sSDKsample = "simpleGL (VBO)";

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void simple_vbo_kernel(float4 *pos, unsigned int width, unsigned int height, float time, float UnitOfChangeOnX, float UnitOfChangeOnY, float4 *offsets, bool falling, float horizontalChange, float randomHeightTop, float randomHeightBottom)//, float FallingDistence)//, float jitterAmmountFloat)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	//printf("%d \n", x);
    // calculate uv coordinates
    float u = x / (float) width;
    float v = y / (float) height;
	u = u * 2.0f - 1.0f;//17/10/19 test - making easier to see dots to try make circle (old code - *2.0f - 1.0f;)
	v = v * 2.0f - 1.0f;//*2.0f - 1.0f;
	//16/10/19 test - Q a. start
	if(u > -0.11f + UnitOfChangeOnX & u < 0.11f + UnitOfChangeOnX){// u > -0.11f & u < 0.11f (new reduced x values)
		if(v > -0.125f + UnitOfChangeOnY & v < 0.125f + UnitOfChangeOnY){
			pos[y*width + x] = make_float4(u, 0.0f, v, 1.0f);
		}
		else {
			float freq = 4.0f;
			float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

			if (falling) {
				if(u > 0.75f & u < 1.0f){//bottom block
					if(v > 0.90f - randomHeightBottom & v < 1.0f){
						//printf("in if");
						//float gameTime = time;
						pos[y*width + x] = make_float4(u-horizontalChange, 0.0f, v, 1.0f);
					}
					else {
						pos[y*width + x] = make_float4(u, -0.5f, v, 1.0f);
					}
				}
				else {
					pos[y*width + x] = make_float4(u, -0.5f, v, 1.0f);
				}
				
				if (u > 0.75f & u < 1.0f) //{//top block
					if (v > -1.1f & v < -0.90f + randomHeightTop) //{
						//printf("in if");
						//float gameTime = time;
						pos[y*width + x] = make_float4(u - horizontalChange, 0.0f, v, 1.0f);
					//}
					//else {
					//	pos[y*width + x] = make_float4(u, -0.5f, v, 1.0f);
					//}
				//}
				//else {
				//	pos[y*width + x] = make_float4(u, -0.5f, v, 1.0f);
				//}
				
			}
			else {
				// write output vertex
				pos[y*width + x] = make_float4(u + offsets[y*width + x].x, w + offsets[y*width + x].y, v + offsets[y*width + x].z, 1.0f);
			}
		}
	}
	else {
		//printf("*** IN ELSE *** \n");
		// calculate simple sine wave pattern
		float freq = 4.0f;
		float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;
		if(falling){
			if(u > 0.75f & u < 1.0f){//bottom block
				if(v > 0.90f - randomHeightBottom & v < 1.0f){
					//printf("in if");
					//float gameTime = time;
					pos[y*width + x] = make_float4(u-horizontalChange, 0.0f, v, 1.0f);
				}
				else {
					pos[y*width + x] = make_float4(u, -0.5f, v, 1.0f);
				}
			}
			else {
				pos[y*width + x] = make_float4(u, -0.5f, v, 1.0f);
			}
			
			if (u > 0.75f & u < 1.0f) //{//top block
				if (v > -1.1f & v < -0.90f + randomHeightTop) //{
					//printf("in if");
					//float gameTime = time;
					pos[y*width + x] = make_float4(u - horizontalChange, 0.0f, v, 1.0f);
				//}
				//else {
				//	pos[y*width + x] = make_float4(u, -0.5f, v, 1.0f);
				//}
			//}
			//else {
			//	pos[y*width + x] = make_float4(u, -0.5f, v, 1.0f);
			//}
			
		}
		else {
			// write output vertex
			pos[y*width + x] = make_float4(u + offsets[y*width + x].x, w + offsets[y*width + x].y, v + offsets[y*width + x].z, 1.0f);
		}
	}
	//16/10/19 test - Q a. end
	__syncthreads();
	//17/10/19 test - Q a. start
	//__syncthreads();//extra top part
	//if (u > -0.109f & u < 0.109f)
	//	if (v > -0.126f & v < 0.126f)
	//		pos[y*width + x] = make_float4(u, 0.0f, v, 1.0f);
	
	if (u > -0.111f + UnitOfChangeOnX & u < 0.111f + UnitOfChangeOnX)//1 - u > -0.111f & u < 0.111f (new reduced x values)
		if (v > -0.109f + UnitOfChangeOnY & v < 0.109f + UnitOfChangeOnY)
			pos[y*width + x] = make_float4(u, 0.0f, v, 1.0f);
	//__syncthreads();
	//if (u > -0.126f & u < 0.126f)//test
	//	if (v > -0.125f & v < 0.125f)
	//		pos[y*width + x] = make_float4(u, 0.0f, v, 1.0f);

	__syncthreads();//2
	if (u > -0.127f + UnitOfChangeOnX & u < 0.127f + UnitOfChangeOnX)// u > -0.127f & u < 0.127f (new reduced x values)
		if (v > -0.093f + UnitOfChangeOnY & v < 0.093f + UnitOfChangeOnY)
			pos[y*width + x] = make_float4(u, 0.0f, v, 1.0f);

	__syncthreads();//3
	if (u > -0.143f + UnitOfChangeOnX & u < 0.143f + UnitOfChangeOnX)// u > -0.143f & u < 0.143f (new reduced x values)
		if (v > -0.077f + UnitOfChangeOnY & v < 0.077f + UnitOfChangeOnY)
			pos[y*width + x] = make_float4(u, 0.0f, v, 1.0f);

	__syncthreads();//4
	if (u > -0.148f + UnitOfChangeOnX & u < 0.148f + UnitOfChangeOnX)// u > -0.148f & u < 0.148f (new reduced x values)
		if (v > -0.061f + UnitOfChangeOnY & v < 0.061f + UnitOfChangeOnY)
			pos[y*width + x] = make_float4(u, 0.0f, v, 1.0f);

	__syncthreads();//5
	if (u > -0.164f + UnitOfChangeOnX & u < 0.164f + UnitOfChangeOnX)// u > -0.164f & u < 0.164f (new reduced x values)
		if (v > -0.045f + UnitOfChangeOnY & v < 0.045f + UnitOfChangeOnY)
			pos[y*width + x] = make_float4(u, 0.0f, v, 1.0f);
	
	__syncthreads();// "top part" 1
	if (u > -0.094f + UnitOfChangeOnX & u < 0.094f + UnitOfChangeOnX)
		if (v > -0.141f + UnitOfChangeOnY & v < 0.141f + UnitOfChangeOnY)
			pos[y*width + x] = make_float4(u, 0.0f, v, 1.0f);

	__syncthreads();// "top part" 2
	if (u > -0.078f + UnitOfChangeOnX & u < 0.078f + UnitOfChangeOnX)
		if (v > -0.157f + UnitOfChangeOnY & v < 0.157f + UnitOfChangeOnY)
			pos[y*width + x] = make_float4(u, 0.0f, v, 1.0f);

	__syncthreads();// "top part" 3
	if (u > -0.062f + UnitOfChangeOnX & u < 0.062f + UnitOfChangeOnX)
		if (v > -0.173f + UnitOfChangeOnY& v < 0.173f + UnitOfChangeOnY)
			pos[y*width + x] = make_float4(u, 0.0f, v, 1.0f);
	
	//17/10/19 test - Q a. end
}

__global__ void new_vbo_x_kernel(float4 *pos, unsigned int width, unsigned int height, float time, float jitterAmmountFloatOrigional)//, float4 *randNum)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	//printf("JitterAmmount: %f\n", jitterAmmountFloat);
	//printf("")
	/*
	float u = x / (float)width;
	float v = y / (float)height;
	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;
	//pos[y*width + x] = make_float4(u + jitterAmmountFloat, 0.0f, v + jitterAmmountFloat, 1.0f);
	*/

	//pos[y*width + x].x = u + jitterAmmountFloat;//make_float4(u + jitterAmmountFloat, 0.0f + jitterAmmountFloat, v + jitterAmmountFloat, 1.0f);
	pos[y*width + x].x += pos[y*width + x].x * jitterAmmountFloatOrigional;
	//pos[y*width + x].y += pos[y*width + x].y * jitterAmmountFloat;
	//pos[y*width + x].z += pos[y*width + x].z * jitterAmmountFloat;
	
}

__global__ void new_vbo_y_kernel(float4 *pos, unsigned int width, unsigned int height, float time, float jitterAmmountFloatOrigional)//, float4 *randNum)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	/*
	float u = x / (float)width;
	float v = y / (float)height;
	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;
	//pos[y*width + x].y = u + jitterAmmountFloat;
	*/
	/*
	float freq = 4.0f;
	float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;
	pos[y*width + x].y = w + jitterAmmountFloat;//make_float4(u, w + jitterAmmountFloat, v, 1.0f);
	*/
	
	pos[y*width + x].y += pos[y*width + x].y * jitterAmmountFloatOrigional;
}

__global__ void new_vbo_z_kernel(float4 *pos, unsigned int width, unsigned int height, float time, float jitterAmmountFloatOrigional)//, float4 *randNum)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	/*
	float u = x / (float)width;
	float v = y / (float)height;
	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;
	pos[y*width + x].z = v + jitterAmmountFloat;
	*/
	pos[y*width + x].z += pos[y*width + x].z * jitterAmmountFloatOrigional;
}

__global__ void game_kernel(float4 *pos, unsigned int width, unsigned int height, float time, bool falling, float UnitOfChangeOnX, float UnitOfChangeOnY)//, float jitterAmmountFloatOrigional)//, float seedTest)//, float4 *randNum)
{
	/*
	if (falling) {
		unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
		unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
		float u = x / (float)width;
		float v = y / (float)height;
		u = u * 2.0f - 1.0f;
		v = v * 2.0f - 1.0f;

		if (u > 0.0f & u < 1.0f)
			if (v > 0.0f & v < -1.0f)
				pos[y*width + x] = make_float4(u, 0.0f, v, 1.0f);

	}
	*/
}

void launch_kernel(float4 *pos, unsigned int mesh_width,
                   unsigned int mesh_height, float time, float4 *offsets)//, float4 *randNum)
{
    // execute the kernel
	
	dim3 block(8, 8, 1);
	dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);

	auto seedTest = std::chrono::high_resolution_clock::now().time_since_epoch().count();
	std::mt19937 generatorTest;
	generatorTest.seed(seedTest);
	std::uniform_real_distribution<double> distributionTest(0.0, 0.75);
	auto seedTest2 = std::chrono::high_resolution_clock::now().time_since_epoch().count();
	std::mt19937 generatorTest2;
	generatorTest2.seed(seedTest2);

	if (falling)
		UnitOfChangeOnY += 0.016;
	if (falling & UnitOfChangeOnY > 1.0f)
		falling = false;//UnitOfChangeOnY = 0.0f;
	if (falling)
		horizontalChange += 0.016f;
	if (falling & horizontalChange > 2.0f)
		horizontalChange = 0.0f;
	if (horizontalChange == 0.0f)
		randomHeightTop = distributionTest(generatorTest);
	if (horizontalChange == 0.0f)
		randomHeightBottom = distributionTest(generatorTest2);
	//auto seedTest = std::chrono::high_resolution_clock::now().time_since_epoch().count();
	game_kernel << < grid, block >> > (pos, mesh_width, mesh_height, time, falling, UnitOfChangeOnX, UnitOfChangeOnY);

	simple_vbo_kernel << < grid, block >> > (pos, mesh_width, mesh_height, time, UnitOfChangeOnX, UnitOfChangeOnY, offsets,falling, horizontalChange, randomHeightTop, randomHeightBottom);//, fallingDistence);//, jitterAmmountFloat);
	//20/10/19 test - jitter
	//if (exitTest != true) {
	    
		if (origionalJitter) {
			//srand(static_cast<unsigned int>(clock()));
			//jitterAmmountFloat = float(rand()) / (float(RAND_MAX)  * 2.0f - 1.0f);//+ 1.0);

			//std::default_random_engine generator;
			//milliseconds ms = duration_cast<milliseconds>(system_clock::now().time_since_epoch());
			//float testFloat = (float)ms;

			auto seed = std::chrono::high_resolution_clock::now().time_since_epoch().count();
			std::mt19937 generator;
			generator.seed(seed);//std::time(0));// * 1000);
			std::uniform_real_distribution<double> distribution(-0.0625 , 0.0652);//-0.03125f, 0.03125f); //doubles from 
			jitterAmmountFloatOrigional = distribution(generator);
			//printf("JitterAmmount x: %f\n", jitterAmmountFloatOrigional);
			new_vbo_x_kernel << <grid, block >> > (pos, mesh_width, mesh_height, time, jitterAmmountFloatOrigional);//, randNum);

			auto seed2 = std::chrono::high_resolution_clock::now().time_since_epoch().count();
			generator.seed(seed2);//std::time(0));// * 1000); 
			jitterAmmountFloatOrigional = distribution(generator);
			//printf("JitterAmmount y: %f\n", jitterAmmountFloatOrigional);
			new_vbo_y_kernel << <grid, block >> > (pos, mesh_width, mesh_height, time, jitterAmmountFloatOrigional);

			auto seed3 = std::chrono::high_resolution_clock::now().time_since_epoch().count();
			generator.seed(seed3);//std::time(0));// * 1000); 
			jitterAmmountFloatOrigional = distribution(generator);
			//printf("JitterAmmount z: %f\n", jitterAmmountFloatOrigional);
			new_vbo_z_kernel << <grid, block >> > (pos, mesh_width, mesh_height, time, jitterAmmountFloatOrigional);
		}
		
	//}
}

bool checkHW(char *name, const char *gpuType, int dev)
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    strcpy(name, deviceProp.name);

    if (!STRNCASECMP(deviceProp.name, gpuType, strlen(gpuType)))
    {
        return true;
    }
    else
    {
        return false;
    }
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    char *ref_file = NULL;

    pArgc = &argc;
    pArgv = argv;

#if defined(__linux__)
    setenv ("DISPLAY", ":0", 0);
#endif

    printf("%s starting...\n", sSDKsample);

    if (argc > 1)
    {
        if (checkCmdLineFlag(argc, (const char **)argv, "file"))
        {
            // In this mode, we are running non-OpenGL and doing a compare of the VBO was generated correctly
            getCmdLineArgumentString(argc, (const char **)argv, "file", (char **)&ref_file);
        }
    }

    printf("\n");

    runTest(argc, argv, ref_file);

    printf("%s completed, returned %s\n", sSDKsample, (g_TotalErrors == 0) ? "OK" : "ERROR!");
    exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

void computeFPS()
{
    frameCount++;
    fpsCount++;

    if (fpsCount == fpsLimit)
    {
        avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
        fpsCount = 0;
        fpsLimit = (int)MAX(avgFPS, 1.f);

        sdkResetTimer(&timer);
    }

    char fps[256];
    sprintf(fps, "Cuda GL Interop (VBO): %3.1f fps (Max 100Hz)", avgFPS);
    glutSetWindowTitle(fps);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int *argc, char **argv)
{
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(window_width, window_height);
    glutCreateWindow("Cuda GL Interop (VBO)");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMotionFunc(motion);
    glutTimerFunc(REFRESH_DELAY, timerEvent,0);

    // initialize necessary OpenGL extensions
    if (! isGLVersionSupported(2,0))
    {
        fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush(stderr);
        return false;
    }

    // default initialization
    glClearColor(0.0, 0.0, 0.0, 1.0);
    glDisable(GL_DEPTH_TEST);

    // viewport
    glViewport(0, 0, window_width, window_height);

    // projection
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)window_width / (GLfloat) window_height, 0.1, 10.0);

    SDK_CHECK_ERROR_GL();

    return true;
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
bool runTest(int argc, char **argv, char *ref_file)
{
    // Create the CUTIL timer
    sdkCreateTimer(&timer);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char **)argv);

    // command line mode only
    if (ref_file != NULL)
    {
        // create VBO
        checkCudaErrors(hipMalloc((void **)&d_vbo_buffer, mesh_width*mesh_height*4*sizeof(float)));

        // run the cuda part
        runAutoTest(devID, argv, ref_file);

        // check result of Cuda step
        checkResultCuda(argc, argv, vbo);

        hipFree(d_vbo_buffer);
        d_vbo_buffer = NULL;
    }
    else
    {
        // First initialize OpenGL context, so we can properly set the GL for CUDA.
        // This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
        if (false == initGL(&argc, argv))
        {
            return false;
        }

        // register callbacks
        glutDisplayFunc(display);
        glutKeyboardFunc(keyboard);
        glutMouseFunc(mouse);
        glutMotionFunc(motion);
#if defined (__APPLE__) || defined(MACOSX)
        atexit(cleanup);
#else
        glutCloseFunc(cleanup);
#endif

        // create VBO
        createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

		//22/10/19 - start
		unsigned int size = mesh_width * mesh_height * 4 * sizeof(float);//1.
		//float4 *h_offsets;//2. MIGHT HAVE TO MAKE THESE GLOBALS
		//float4 *d_offsets;// MIGHT HAVE TO MAKE THESE GLOBALS
		//float4 **testh = &h_offsets;
		//float4 **testd = &d_offsets;
		h_offsets = (float4*)malloc(size);//test - size);//3
		//hipMalloc(&d_offsets, size);//test - size);
		hipMalloc((void **)&d_offsets, size);//exactly lab3 way

		//for (int i = 0; i < size; i++) { h_offsets[i] = { 0.1f, 0.1f, 0.1f, 0.1f }; }//4 - fill h_offset with random xyz THESE MIGHT HAVE TO HAPPEN IN RUNCUDA

		//hipMemcpy(d_offsets, h_offsets, size, hipMemcpyHostToDevice);//5 THESE MIGHT HAVE TO HAPPEN IN RUNCUDA
		/*
		if (jitter) {
			for (int i = 0; i < size; i++) { h_offsets[i] = make_float4(0.5f, 0.5f, 0.5f, 1.0f); }//4
		}
		else {
			for (int i = 0; i < 20000; i++) { h_offsets[i] = make_float4(0.0f, 0.0f, 0.0f, 1.0f); }//4
		}
		hipMemcpy(d_offsets, h_offsets, size, hipMemcpyHostToDevice);//5
		*/
		//22/10/19 - end

		//20/10/19
		//createVBO2(&vbo2, &cuda_vbo_resource_2, cudaGraphicsMapFlagsWriteDiscard);

        // run the cuda part
		runCuda(&cuda_vbo_resource, &h_offsets, &d_offsets);//, &cuda_vbo_resource_2);//, 0);// 18/10/19 test - UI

        // start rendering mainloop
        glutMainLoop();
    }

    return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource **vbo_resource, float4 **Hoffsets, float4 **Doffsets)//, struct hipGraphicsResource **vbo_resource_2)// 18/10/19 test - UI
{
	
		// map OpenGL buffer object for writing from CUDA
		float4 *dptr;
		checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
		size_t num_bytes;
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
			*vbo_resource));
		//printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);

		// execute the kernel
		//    dim3 block(8, 8, 1);
		//    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
		//    kernel<<< grid, block>>>(dptr, mesh_width, mesh_height, g_fAnim);

		/*
		//20/10/19
		float4 *dptr2;
		checkCudaErrors(hipGraphicsMapResources(1, vbo_resource_2, 0));
		size_t num_bytes_2;
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr2, &num_bytes_2,
			*vbo_resource_2));
		*/
		//22/10/19 test - start
		
		float4 *dptr2;//23/10/19
		float4 *dptr3;//23/10/19
		dptr2 = *Hoffsets;
		dptr3 = *Doffsets;
		unsigned int size = mesh_width * mesh_height * 4 * sizeof(float);
		if (jitter) {
			auto seed = std::chrono::high_resolution_clock::now().time_since_epoch().count();
			std::mt19937 generator;
			generator.seed(seed);
			std::uniform_real_distribution<double> distribution(-0.0625, 0.0652);
			jitterAmmountFloat1 = distribution(generator);
			//printf("JitterAmmountFloat1: %f\n", jitterAmmountFloat1);

			auto seed2 = std::chrono::high_resolution_clock::now().time_since_epoch().count();
			generator.seed(seed2);
			jitterAmmountFloat2 = distribution(generator);
			//printf("JitterAmmountFloat2: %f\n", jitterAmmountFloat2);

			auto seed3 = std::chrono::high_resolution_clock::now().time_since_epoch().count();
			generator.seed(seed3);
			jitterAmmountFloat3 = distribution(generator);
			//printf("JitterAmmountFloat3: %f\n", jitterAmmountFloat3);
			for (int i = 0; i < 60000; i++) { dptr2[i] = make_float4(jitterAmmountFloat1, jitterAmmountFloat2, jitterAmmountFloat3, 1.0f); }//4
		}
		else {
			for (int i = 0; i < 60000; i++) { dptr2[i] = make_float4(0.0f, 0.0f, 0.0f, 1.0f); }//4
		}
		hipMemcpy(dptr3, dptr2, 60000, hipMemcpyHostToDevice);//5
		
		//22/10/19 test - end
		launch_kernel(dptr, mesh_width, mesh_height, g_fAnim, dptr3);//, d_offsets);//, dptr2);

    // unmap buffer object
    checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
	//checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource_2, 0));
}

#ifdef _WIN32
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) fopen_s(&fHandle, filename, mode)
#endif
#else
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) (fHandle = fopen(filename, mode))
#endif
#endif

void sdkDumpBin2(void *data, unsigned int bytes, const char *filename)
{
    printf("sdkDumpBin: <%s>\n", filename);
    FILE *fp;
    FOPEN(fp, filename, "wb");
    fwrite(data, bytes, 1, fp);
    fflush(fp);
    fclose(fp);
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runAutoTest(int devID, char **argv, char *ref_file)
{
    char *reference_file = NULL;
    void *imageData = malloc(mesh_width*mesh_height*sizeof(float));
	//char ui = ' '; 18/10/19 test - UI
    // execute the kernel
	launch_kernel((float4 *)d_vbo_buffer, mesh_width, mesh_height, g_fAnim, (float4 *)offsetsAutoTestD);//, d_offsets);//, (float4 *)d_vbo_buffer_2);//, 0);

    hipDeviceSynchronize();
    getLastCudaError("launch_kernel failed");

    checkCudaErrors(hipMemcpy(imageData, d_vbo_buffer, mesh_width*mesh_height*sizeof(float), hipMemcpyDeviceToHost));

    sdkDumpBin2(imageData, mesh_width*mesh_height*sizeof(float), "simpleGL.bin");
    reference_file = sdkFindFilePath(ref_file, argv[0]);

    if (reference_file &&
        !sdkCompareBin2BinFloat("simpleGL.bin", reference_file,
                                mesh_width*mesh_height*sizeof(float),
                                MAX_EPSILON_ERROR, THRESHOLD, pArgv[0]))
    {
        g_TotalErrors++;
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags)
{
    assert(vbo);

    // create buffer object
    glGenBuffers(1, vbo);
    glBindBuffer(GL_ARRAY_BUFFER, *vbo);

    // initialize buffer object
    unsigned int size = mesh_width * mesh_height * 4 * sizeof(float);
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // register this buffer object with CUDA
    checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

    SDK_CHECK_ERROR_GL();
}
/*
void createVBO2(GLuint *vbo2, struct hipGraphicsResource **vbo_res_2,
	unsigned int vbo_res_flags_2)
{
	assert(vbo2);

	// create buffer object
	glGenBuffers(1, vbo2);
	glBindBuffer(GL_ARRAY_BUFFER, *vbo2);

	// initialize buffer object
	unsigned int size = ((float(rand()) / float(RAND_MAX)) * (1.0f - -1.0f)) + -1.0f;
	glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

	glBindBuffer(GL_ARRAY_BUFFER, 1);

	// register this buffer object with CUDA
	checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res_2, *vbo2, vbo_res_flags_2));

	SDK_CHECK_ERROR_GL();
}
*/
////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{

    // unregister this buffer object with CUDA
    checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

    glBindBuffer(1, *vbo);
    glDeleteBuffers(1, vbo);

    *vbo = 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
    sdkStartTimer(&timer);
    // run CUDA kernel to generate vertex positions
	runCuda(&cuda_vbo_resource, &h_offsets, &d_offsets);//, h_offsets, d_offsets);//, &cuda_vbo_resource_2);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(0.0, 0.0, translate_z);
    glRotatef(rotate_x, 1.0, 0.0, 0.0);
    glRotatef(rotate_y, 0.0, 1.0, 0.0);

    // render from the vbo
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glVertexPointer(4, GL_FLOAT, 0, 0);

    glEnableClientState(GL_VERTEX_ARRAY);
    glColor3f(1.0, 0.0, 0.0);
    glDrawArrays(GL_POINTS, 0, mesh_width * mesh_height);
    glDisableClientState(GL_VERTEX_ARRAY);

    glutSwapBuffers();

    g_fAnim += 0.01f;

    sdkStopTimer(&timer);
    computeFPS();
}

void timerEvent(int value)
{
    if (glutGetWindow())
    {
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, timerEvent,0);
    }
}

void cleanup()
{
    sdkDeleteTimer(&timer);

    if (vbo)
    {
        deleteVBO(&vbo, cuda_vbo_resource);
		//deleteVBO(&vbo2, cuda_vbo_resource_2);//20/10/19
    }
}


////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
	//float4 * dptr;//18/10/19 test - UI
    switch (key)
    {
        case (27) :
            #if defined(__APPLE__) || defined(MACOSX)
                exit(EXIT_SUCCESS);
            #else
                glutDestroyWindow(glutGetWindow());
                return;
            #endif

		//18/10/19 test - UI
		case (115) :
			//launch_new_kernel((float4 *)d_vbo_buffer, mesh_width, mesh_height, g_fAnim, 1);
			//runCuda(&cuda_vbo_resource, 1);
			if(!falling)
				if(UnitOfChangeOnY < 1.072f)
					UnitOfChangeOnY += 0.032f;// 19/10/19 test - UI
			//display();// 19/10/19 test - UI
			printf("JUST_PRESSED_s \n");
			return;

		case (119):
			if (UnitOfChangeOnY > -1.072f)
				UnitOfChangeOnY -= 0.032f;

			if(falling)
				if(UnitOfChangeOnY > -1.072f)
					UnitOfChangeOnY -= 0.256f;

			printf("JUST_PRESSED_w \n");
			return;

		case (97):
			if (!falling)
				if (UnitOfChangeOnX > -1.072f)
					UnitOfChangeOnX -= 0.032f;
			printf("JUST_PRESSED_a \n");
			return;

		case (100):
			if (!falling)
				if (UnitOfChangeOnX < 1.072f)
					UnitOfChangeOnX += 0.032f;
			printf("JUST_PRESSED_d \n");
			return;

		case (49):
			//if (exitTest)
			//	exitTest = false;
			jitter = true;
			//srand(static_cast<unsigned int>(clock()));
			//jitterAmmountInt = 0 + (rand() %2);//(rand() / (float)RAND_MAX * 1.0f) + -1.0f;//((float(rand()) / float(RAND_MAX)) * 1.0f - -1.0f);//(1.0f - -1.0f)) + -1.0f;//* (1.0f - -1.0f)) + -1.0f;
			//jitterAmmountFloat = double(rand()) / (double(RAND_MAX) + 1.0);//float(5 + rand() % (150 +1 -5))/ 100;
			//printf("JitterAmmountInt: %d\n", jitterAmmountInt);
			//printf("JitterAmmountFloat: %f\n", jitterAmmountFloat);
			printf("JUST_PRESSED_1 \n");
			return;

		case (50):
			//exitTest = true;
			jitter = false;
			printf("JUST_PRESSED_2 \n");
			return;

		case (53):
			falling = true;
			UnitOfChangeOnX = 0.0f;
			UnitOfChangeOnY = 0.0f;
			/*
			while (falling)
			{
				if (fallingDistence > 1.0f)
					fallingDistence = 0.0f;

				fallingDistence += 0.016f;
			}
			*/
			printf("JUST_PRESSED_5 \n");
			return;

		case(54):
			falling = false;
			//fallingDistence = 0.0f;
			printf("JUST_PRESSED_6 \n");
			return;

		case(51):
			origionalJitter = true;
			printf("JUST_PRESSED_3 \n");
			return;

		case(52):
			origionalJitter = false;
			printf("JUST_PRESSED_4 \n");
			return;
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
    {
        mouse_buttons |= 1<<button;
    }
    else if (state == GLUT_UP)
    {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void motion(int x, int y)
{
    float dx, dy;
    dx = (float)(x - mouse_old_x);
    dy = (float)(y - mouse_old_y);

    if (mouse_buttons & 1)
    {
        rotate_x += dy * 0.2f;
        rotate_y += dx * 0.2f;
    }
    else if (mouse_buttons & 4)
    {
        translate_z += dy * 0.01f;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

////////////////////////////////////////////////////////////////////////////////
//! Check if the result is correct or write data to file for external
//! regression testing
////////////////////////////////////////////////////////////////////////////////
void checkResultCuda(int argc, char **argv, const GLuint &vbo)
{
    if (!d_vbo_buffer)
    {
        checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));

        // map buffer object
        glBindBuffer(GL_ARRAY_BUFFER, vbo);
        float *data = (float *) glMapBuffer(GL_ARRAY_BUFFER, GL_READ_ONLY);

        // check result
        if (checkCmdLineFlag(argc, (const char **) argv, "regression"))
        {
            // write file for regression test
            sdkWriteFile<float>("./data/regression.dat",
                                data, mesh_width * mesh_height * 3, 0.0, false);
        }

        // unmap GL buffer object
        if (!glUnmapBuffer(GL_ARRAY_BUFFER))
        {
            fprintf(stderr, "Unmap buffer failed.\n");
            fflush(stderr);
        }

        checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, vbo,
                                                     cudaGraphicsMapFlagsWriteDiscard));

        SDK_CHECK_ERROR_GL();
    }
}
